
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdlib.h>

using namespace std;
__global__ void print(unsigned long long * a){
    // printf("1234567\n");
    for(int i = 0; i < 8; i++){
        printf("%llu\t",a[i]);
    }printf("\n");
}
struct uint128_t{
    unsigned long long low;
    unsigned long long high;
};
__device__ __host__ __forceinline__ void bigMul(unsigned long long a,unsigned long long b, uint128_t& res){
    u_int64_t a0 = a & 0xffffffff;
    u_int64_t a1 = a >> 32;
    u_int64_t b0 = b & 0xffffffff;
    u_int64_t b1 = b >> 32;
    u_int64_t low = a0 * b0 ;
    u_int64_t carry = 0;
    if(low > low + ((a0 * b1) << 32llu)){
        carry+=1;
    }
    low = low + ((a0 * b1) << 32llu);
    if(low > low + ((a1 * b0) << 32llu)){
        carry+=1;
    }
    low = low + ((a1 * b0) << 32llu);

    u_int64_t high = a1 * b1 + ((a0 * b1)>> 32llu) + ((a1 * b0)>> 32llu) + carry;
    // high += ((a0 * b0) >> 32) + 
    res.low = low;
    res.high = high;
    // return res;
}
__device__ __host__ __forceinline__ void bigIntegerMul(unsigned long long * a, unsigned long long b,int size){
    uint128_t temp; 
    bigMul(a[0],b,temp);
    // printf("%llu,%llu\n",temp.low,temp.high);
    unsigned long long carry = temp.high;
    a[0] = temp.low;
    // printf("%llu,%llu\n",a[0],temp.low);
    for(int i = 1; i < size; i++){
        if(!a[i] && !carry){
            break;
        }
        uint128_t temp;
        bigMul(a[i],b,temp);
        // temp = te
        a[i] = temp.low;
        // a[i] = (temp & 0xffffffffffffffff);
        if(a[i] + carry < a[i]){
            a[i] += carry;
            carry = 1 + temp.high;
        }
        else{
            a[i] += carry;
            carry = temp.high;
        }
    }
}
__device__ __host__ __forceinline__ void bigIntegerMul_d(unsigned long long * a, unsigned long long b,int size){
    uint128_t temp;
    bigMul(a[0],b,temp);
    
    printf("%llu,%llu,%llu,%llu\n",a[0],b,temp.low,temp.high);
    unsigned long long carry = temp.high;
    // printf("%llu,%llu\t",a[0],temp.low);
    a[0] = temp.low;
    // printf("%llu,%llu\n",a[0],temp.low);
    for(int i = 1; i < size; i++){
        if(!a[i] && !carry){
            break;
        }
        uint128_t temp;
        bigMul(a[i],b,temp);
        // temp = te
        a[i] = temp.low;
        // a[i] = (temp & 0xffffffffffffffff);
        if(a[i] + carry < a[i]){
            a[i] += carry;
            carry = 1 + temp.high;
        }
        else{
            a[i] += carry;
            carry = temp.high;
        }
    }
}
__device__ __host__ __forceinline__ void bigIntegerAdd(unsigned long long * a, unsigned long long *b,int size){
    unsigned long long carry = 0;
    for(int i = 0; i < size; i++){
        if(a[i] + b[i] + carry < a[i]){
            // carry = 1;
            a[i] = a[i] + b[i] + carry;
            carry = 1;
        }
        else{
            a[i] = a[i] + b[i] + carry;
            carry = 0;
        }
    }
}
__device__ __host__ __forceinline__ int isneg(unsigned long long * a, unsigned long long *b ,unsigned long long p,int size){
    // unsigned long long* temp = (unsigned long long*)malloc(size * sizeof(unsigned long long)); 
    unsigned long long temp[64];
    for(int i = 0; i < size; i++){
        temp[i] = b[i];
    }
    bigIntegerMul(temp,p,size);
    for(int i = size - 1; i > 0; i--){
        if(a[i] > temp[i]){
            // free(temp);
            return 1;
        }
        else if(a[i] < temp[i]){
            // free(temp);
            return -1;
        }
    }
    // free(temp);
    return 0;
}
__device__ __host__ __forceinline__ unsigned long long bigIntegerMod(unsigned long long * a, unsigned long long *b,int size){
    unsigned long long l = 0;
    unsigned long long r = (1llu << 63);
    unsigned long long guess;
        // if(a[0] == 16230973104951096732){
        //     printf("%p\n",a);
        // }
    while(l <= r){

        // printf("%llu,%llu\n",l,r);
        unsigned long long guess = (l + r)/2;
        int res = isneg(a,b,guess,size);
        if(res == 1){
            l = guess + 1;
        }
        else if(res == -1){
            r = guess;
        }
        else{
            return a[0] - (b[0] * guess);
        }
    }
    return 0;
}
__global__ void cudadecompose(unsigned long long *list,unsigned long long* moduleChain,int listLen,int moduleLen,unsigned long long * decomposeList){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    for(int i = 0; i < moduleLen; i++){
        decomposeList[i * listLen + tid] = list[tid] % moduleChain[i];
    }
    // if(tid == 1){
    //     printf("**%llu\n",list[1]);
    // }
}
__global__ void cudacompose(unsigned long long *decomposeList,
                            unsigned long long* moduleChain,
                            int listLen,
                            int moduleLen,
                            unsigned long long* Ni,
                            unsigned long long *bigN,unsigned long long * composeList,unsigned long long* temp1,unsigned long long* temp2){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // unsigned long long temp1[64];
    // unsigned long long temp2[64];
    for(int i = 0; i < moduleLen; i++){
        temp2[tid * moduleLen + i] = 0;
    }
    for(int i = 0; i < moduleLen; i++){
        for(int j = 0; j < moduleLen; j++){
            temp1[tid * moduleLen + j] = Ni[i * moduleLen + j];
        }
        
    // if(i==0)printf("%llu,%llu\n",decomposeList[i * listLen + tid],temp1[tid * moduleLen]);
        bigIntegerMul(&(temp1[tid * moduleLen]),decomposeList[i * listLen + tid],moduleLen);
        // if(i == 0)printf("%llu\n",temp1[tid * moduleLen]);

        bigIntegerAdd(&temp2[tid * moduleLen],&temp1[tid * moduleLen],moduleLen);
    }
    // if(tid == 114){
    //     printf("%llu\n",decomposeList[i * listLen + tid]);
    //     // printf("%llu\t",Ni[0]);
    //     for(int i = 0; i < 8; i++){
    //         printf("%llu\t",temp1[i]);
    //     }printf("\n");
    //     // for(int i = 0; i < 8; i++){
    //     //     printf("%llu\t",bigN[i]);
    //     // }printf("\n");
    // }
    // }
    // if(tid == 514){
    //     // printf("%llu\t",Ni[0]);
    //     // printf("%p\n",temp1);
    //     for(int i = 0; i < 8; i++){
    //         printf("%llu\t",temp2[tid * moduleLen + i]);
    //     }printf("\n");
    //     for(int i = 0; i < 8; i++){
    //         printf("%llu\t",bigN[i]);
    //     }printf("\n");
    // }
    unsigned long long res = bigIntegerMod(&temp2[tid * moduleLen],bigN,moduleLen);
    composeList[tid] = res;
}


class RNS{
    public:
    int N;
    unsigned long long* moduleChain;
    unsigned long long* Ni;
    unsigned long long *bigN;
    unsigned long long *buff1;
    unsigned long long *buff2;
    RNS(int N,unsigned long long scale){
        this->N = N;
        unsigned long long* moduleChain_h = (unsigned long long*)malloc(N * sizeof(unsigned long long));
        genPrime(moduleChain_h,scale,N);
        // for(int i = 0; i < N; i++){
        //     printf("%llu\t",moduleChain_h[i]);
        // }printf("\n");
        unsigned long long** Ni_h = (unsigned long long**)calloc(N,sizeof(unsigned long long**));
        for(int i = 0; i < N;i++){
            Ni_h[i] = (unsigned long long*)calloc(N,sizeof(unsigned long long));
        }
        // unsigned long long bigN[N];
        unsigned long long *bigN_h = (unsigned long long*)calloc(N,sizeof(unsigned long long));
        unsigned long long *ti_h = (unsigned long long*)calloc(N,sizeof(unsigned long long));
        bigN_h[0] = 1;
        for(int i = 0;i < N; i++){
            Ni_h[i][0] = 1;
            ti_h[i] = 1;
        }
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(i==j)continue;
                bigIntegerMul(Ni_h[i],moduleChain_h[j],N);
                ti_h[i] = ti_h[i] * qpow(moduleChain_h[j],moduleChain_h[i]-2,moduleChain_h[i]) % moduleChain_h[i];
            }
        }
        
        for(int i = 0; i < N; i++){
            bigIntegerMul(bigN_h,moduleChain_h[i],N);
        }
        for(int i = 0; i < N; i++){
            bigIntegerMul(Ni_h[i],ti_h[i],N);
        }
        free(ti_h);
        
        hipMalloc(&moduleChain,N * sizeof(unsigned long long));
        hipMalloc(&Ni,N * N * sizeof(unsigned long long));
        hipMalloc(&bigN,N * sizeof(unsigned long long));
        hipMemcpy(moduleChain,moduleChain_h,N * sizeof(unsigned long long),hipMemcpyHostToDevice);
        hipMemcpy(bigN,bigN_h,N * sizeof(unsigned long long),hipMemcpyHostToDevice);
        // for(int i = 0; i < N; i++){
        //     for(int j = 0; j < N;j++){
        //         printf("%llu\t",Ni_h[i][j]);
        //     }
        //     printf("\n");
        // }
        // exit(1);
        for(int i = 0; i < N; i++){
            hipMemcpy(Ni+(N*i),Ni_h[i],N * sizeof(unsigned long long),hipMemcpyHostToDevice);
        }
        print<<<1,1>>>(Ni);
        // exit(1);
    }
    unsigned long long* decompose(unsigned long long *list,int listLen){
        unsigned long long * decomposeList;
        hipMalloc(&decomposeList, listLen * N * sizeof(unsigned long long));
        hipMalloc(&buff1, listLen * N * sizeof(unsigned long long));
        hipMalloc(&buff2, listLen * N * sizeof(unsigned long long));
        cudadecompose<<<listLen/1024,1024>>>(list,moduleChain,listLen,N,decomposeList);
        return decomposeList;
    }
    unsigned long long* compose(unsigned long long * decomposeList, int listLen){
        unsigned long long * composeList;
        hipMalloc(&composeList, listLen * N * sizeof(unsigned long long));
        print<<<1,1>>>(Ni);
        cudacompose<<<listLen/1024,1024>>>(decomposeList,moduleChain,listLen,N,Ni,bigN,composeList,buff1,buff2);
        return composeList;        
    }
    private:
    void genPrime(unsigned long long* moduleChain_h,unsigned long long scale,int N){
        if(scale % 2 == 0){
            scale+=1;
        }
        int cnt = 0;
        while(cnt < N){
            if(MillerRabin(scale)){
                moduleChain_h[cnt++] = scale;
                scale += 2;
            }
            else{
                scale += 2;
            }
        }
    }
    unsigned long long qpow(unsigned long long a,unsigned long long b,unsigned long long q){
        unsigned long long r=1;
        // unsigned long long base = a;
        while(b){
            if(b&1)r = (a * r)%q;
            a = a * a % q;
            b >>= 1;
        }
        return r;
    }
    bool MillerRabin(unsigned long long n){
        if(n == 2){
            return true;
        }
        if(n % 2 == 0){
            return false;
        }
        bool res = true;
        for(unsigned long long a = 2; a < 64 && a < n; a++){
            unsigned long long d = n - 1;
            // printf("%llu\n",a);
            while(d % 2 == 0){
                // printf("%llu,%llu,%llu\n",a,d,qpow(a,d,n));
                if(qpow(a,d,n) == 1){
                    // continue;
                }
                else if(qpow(a,d,n) == n-1){
                    break;
                }
                else{
                    return false;
                }
                d /=2;
            }
        }
        return true;
    }
};

__global__ void init(unsigned long long* a){
    for(int i = 0; i < 1024; i++){
        a[i] = i * 1919;
    }
}

int main(){
    RNS rns(8,10000);
    unsigned long long *ptr;
    hipMalloc(&ptr,1024 * sizeof(unsigned long long));
    init<<<1,1>>>(ptr);
    unsigned long long *res = rns.decompose(ptr,1024);
    printf("decompose finish\n");
    unsigned long long *ori = rns.compose(res,1024);
    print<<<1,1>>>(ori);
    hipDeviceSynchronize();
}