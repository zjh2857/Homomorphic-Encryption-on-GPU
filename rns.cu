
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdlib.h>

using namespace std;
class uint128_t{
    public:

    unsigned long long low;
    unsigned long long high;
    uint128_t(unsigned long long low, unsigned long long high){
        this->low = low;
        this->high = high;
    }
};
__host__ __device__ __forceinline__ uint128_t bigMul(unsigned long long a,unsigned long long b){
    u_int32_t a0 = a;
    u_int32_t a1 = a >> 32;
    u_int32_t b0 = b;
    u_int32_t b1 = b >> 32;
    u_int64_t low = a0 * b0 + ( (a0 * b1 + a1 * b0) & 0xffffffff );
    u_int64_t high = a1 * b1 + ( (a0 * b1 + a1 * b0) >> 32llu );
    uint128_t res(low,high);
    return res;
}
__device__ __host__ void bigIntegerMul(unsigned long long * a, unsigned long long b,int size){
    uint128_t temp = bigMul(a[0],b);
    unsigned long long carry = temp.high;
    
    for(int i = 1; i < size; i++){
        if(!a[i] && !carry){
            break;
        }
        uint128_t temp = bigMul(a[i],b);
        // temp = te
        a[i] = temp.low;
        // a[i] = (temp & 0xffffffffffffffff);
        if(a[i] + carry < a[i]){
            a[i] += carry;
            carry = 1 + temp.high;
        }
        else{
            a[i] += carry;
            carry = 1 + temp.high;
        }
    }
}
__device__ __host__ void bigIntegerAdd(unsigned long long * a, unsigned long long *b,int size){
    unsigned long long carry = 0;
    for(int i = 0; i < size; i++){
        if(a[i] + b[i] + carry < a[i]){
            // carry = 1;
            a[i] = a[i] + b[i] + carry;
            carry = 1;
        }
        else{
            a[i] = a[i] + b[i] + carry;
            carry = 0;
        }
    }
}
__device__ __host__  int isneg(unsigned long long * a, unsigned long long *b ,unsigned long long p,int size){
    // unsigned long long* temp = (unsigned long long*)malloc(size * sizeof(unsigned long long)); 
    unsigned long long temp[64];
    for(int i = 0; i < size; i++){
        temp[i] = b[i];
    }
    bigIntegerMul(temp,p,size);
    for(int i = size - 1; i > 0; i--){
        if(a[i] > temp[i]){
            // free(temp);
            return 1;
        }
        else if(a[i] < temp[i]){
            // free(temp);
            return -1;
        }
    }
    // free(temp);
    return 0;
}
__device__ __host__ unsigned long long bigIntegerMod(unsigned long long * a, unsigned long long *b,int size){
    unsigned long long l = 0;
    unsigned long long r = (1llu << 63);
    unsigned long long guess;
    while(l <= r){
        printf("%llu,%llu\n",l,r);
        unsigned long long guess = (l + r)/2;
        int res = isneg(a,b,guess,size);
        if(res == 1){
            l = guess + 1;
        }
        else if(res == -1){
            r = guess;
        }
        else{
            return a[0] - (b[0] * guess);
        }
    }
    return 0;
}
__global__ void cudadecompose(unsigned long long *list,unsigned long long* moduleChain,int listLen,int moduleLen,unsigned long long * decomposeList){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    for(int i = 0; i < moduleLen; i++){
        decomposeList[i * listLen + tid] = list[tid] % moduleChain[i];
    }
}
__global__ void cudacompose(unsigned long long *decomposeList,
                            unsigned long long* moduleChain,
                            int listLen,
                            int moduleLen,
                            unsigned long long* Ni,
                            unsigned long long *bigN,unsigned long long * composeList){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned long long temp1[64];
    unsigned long long temp2[64];
    for(int i = 0; i < moduleLen; i++){
        temp1[i] = 0;
        temp2[i] = 0;
    }
    temp1[0] = 1;
    for(int i = 0; i < moduleLen; i++){
        bigIntegerMul(temp1,decomposeList[i * listLen + tid],moduleLen);
        bigIntegerAdd(temp2,temp1,moduleLen);
    }
    unsigned long long res = bigIntegerMod(temp2,bigN,moduleLen);
    composeList[tid] = res;
}


class RNS{
    public:
    int N;
    unsigned long long* moduleChain;
    unsigned long long* Ni;
    unsigned long long *bigN;
    RNS(int N,unsigned long long scale){
        this->N = N;
        unsigned long long* moduleChain_h = (unsigned long long*)malloc(N * sizeof(unsigned long long));
        genPrime(moduleChain_h,scale,N);
        unsigned long long** Ni_h = (unsigned long long**)calloc(N,sizeof(unsigned long long**));
        for(int i = 0; i < N;i++){
            Ni_h[i] = (unsigned long long*)calloc(N,sizeof(unsigned long long));
        }
        // unsigned long long bigN[N];
        unsigned long long *bigN_h = (unsigned long long*)calloc(N,sizeof(unsigned long long));
        unsigned long long *ti_h = (unsigned long long*)calloc(N,sizeof(unsigned long long));
        bigN_h[0] = 1;
        for(int i = 0;i < N; i++){
            Ni_h[i][0] = 1;
            ti_h[i] = 1;
        }
        for(int i = 0; i < N; i++){
            for(int j = 0; j < N; j++){
                if(i==j)continue;
                bigIntegerMul(Ni_h[i],moduleChain[j],N);
                ti_h[i] = ti_h[i] * qpow(moduleChain[j],moduleChain[i]-2,moduleChain[i]) % moduleChain[i];
            }
        }
        
        for(int i = 0; i < N; i++){
            bigIntegerMul(bigN_h,moduleChain_h[i],N);
        }
        for(int i = 0; i < N; i++){
            bigIntegerMul(Ni_h[i],ti_h[i],N);
        }
        free(ti_h);
        
        hipMalloc(&moduleChain,N * sizeof(unsigned long long));
        hipMalloc(&Ni,N * N * sizeof(unsigned long long));
        hipMalloc(&bigN,N * sizeof(unsigned long long));
        hipMemcpy(moduleChain,moduleChain_h,N * sizeof(unsigned long long),hipMemcpyHostToDevice);
        hipMemcpy(bigN,bigN_h,N * sizeof(unsigned long long),hipMemcpyHostToDevice);
        
        for(int i = 0; i < N; i++){
            hipMemcpy(Ni+(N*i),bigN_h,N * sizeof(unsigned long long),hipMemcpyHostToDevice);
        }
    }
    unsigned long long* decompose(unsigned long long *list,int listLen){
        unsigned long long * decomposeList;
        hipMalloc(&decomposeList, listLen * N * sizeof(unsigned long long));
        cudadecompose<<<listLen/1024,1024>>>(list,moduleChain,listLen,N,decomposeList);
        return decomposeList;
    }
    unsigned long long* compose(unsigned long long * decomposeList, int listLen){
        unsigned long long * composeList;
        hipMalloc(&composeList, listLen * N * sizeof(unsigned long long));
        cudacompose<<<listLen/1024,1024>>>(decomposeList,moduleChain,listLen,N,Ni,bigN,composeList);
        return composeList;        
    }
    private:
    void genPrime(unsigned long long* moduleChain_h,unsigned long long scale,int N){
        if(scale % 2 == 0){
            scale+=1;
        }
        int cnt = 0;
        while(cnt < N){
            if(MillerRabin(scale)){
                moduleChain_h[cnt++] = scale;
                scale += 2;
            }
            else{
                scale += 2;
            }
        }
    }
    unsigned long long qpow(unsigned long long a,unsigned long long b,unsigned long long q){
        unsigned long long r=1;
        // unsigned long long base = a;
        while(b){
            if(b&1)r = (a * r)%q;
            a = a * a % q;
            b >>= 1;
        }
        return r;
    }
    bool MillerRabin(unsigned long long n){
        if(n == 2){
            return true;
        }
        if(n % 2 == 0){
            return false;
        }
        bool res = true;
        for(unsigned long long a = 2; a < 64 && a < n; a++){
            unsigned long long d = n - 1;
            // printf("%llu\n",a);
            while(d % 2 == 0){
                // printf("%llu,%llu,%llu\n",a,d,qpow(a,d,n));
                if(qpow(a,d,n) == 1){
                    // continue;
                }
                else if(qpow(a,d,n) == n-1){
                    break;
                }
                else{
                    return false;
                }
                d /=2;
            }
        }
        return true;
    }
};

__global__ void init(unsigned long long* a){
    for(int i = 0; i < 1024; i++){
        a[i] = i * 10000;
    }
}
__global__ void print(unsigned long long * a){
    printf("1234567\n");
    for(int i = 0; i < 8; i++){
        printf("%llu\t");
    }printf("\n");
}
int main(){
    RNS rns(1024,10000);
    unsigned long long *ptr;
    hipMalloc(&ptr,1024 * sizeof(unsigned long long));
    init<<<1,1>>>(ptr);
    unsigned long long *res = rns.decompose(ptr,1024);
    unsigned long long *ori = rns.compose(res,1024);
    printf("%p\n",ori);
    print<<<1,1>>>(ori);
    hipDeviceSynchronize();
}