#include "hip/hip_runtime.h"
#include "encoder.cuh"
#include"hipfft/hipfft.h"
#include "encryptor.cuh"
#include "evaluator.cuh"
#include "freshman.h"
const int testTimes = 100;

__global__ void print(unsigned long long* a){
    for(int i = 0; i < 8;i++){
        printf("%llu,",a[i]);
    }printf("\n");
}
__global__ void print(unsigned long long* a,unsigned long long id){
    printf("%llu\n",id);
    for(int i = 0; i < 8;i++){
        printf("%llu ",a[i]);
    }printf("\n");
}
__global__ void print_d(unsigned long long* a,int d){
    for(int i = 0; i < 1;i++){
        printf("printf_d%d:%llu ",d,a[d]);
    }printf("\n");
}
int main(){
    printf("===\n");
    int N = 2048;
    double scale = 1llu << 30;
    double a[N];
    for(int i = 0; i < N; i++){
        a[i] = i ;
    }
    // a[N/2-1] = 10000;
    // a[0] = 10;
    double b[N];
    for(int i = 0; i < N; i++){
        b[i] = i;
    }

    keyGen keygen(N,scale,8);
    Encoder encoder(N,scale,8);
    Encryptor encryptor(N,scale,8);
    Evaluator evaluator(N,8);
    // double encodeTimes = cpuSecond();
    // for(int i = 0; i < testTimes; i++){
    //     auto foo = encoder.encode(a);
    // }
    // hipDeviceSynchronize();
    // printf("encode Times:%lf microseconds",(cpuSecond() - encodeTimes)/testTimes);
    auto encodeVeca = encoder.encode(a);
    auto encodeVecb = encoder.encode(b);
    double encryptoTimes = cpuSecond();
    for(int i = 0; i < testTimes; i++){
        auto foo = encryptor.encrypt(encodeVeca,keygen.pub);
    }
    hipDeviceSynchronize();
    printf("encrypto Times:%lf seconds",(cpuSecond() - encryptoTimes)/testTimes);
    auto ciptertexta = encryptor.encrypt(encodeVeca,keygen.pub);
    auto ciptertextb = encryptor.encrypt(encodeVecb,keygen.pub);
    auto ciptertextc = evaluator.mulcipter(ciptertexta,ciptertextb);
    auto ciptertextd = evaluator.relien(ciptertextc,keygen.relien);
    evaluator.rescale(ciptertextd);
    unsigned long long* dec = encryptor.decrypt(ciptertextd,keygen.pri);
    auto plaina = encoder.decode(dec);
    for(int i = 0; i < 20; i++){
        printf("%lf\n",plaina[i] * 1179649 / scale);
    }
    //  auto plainb = encoder.decode(encodeVecb);
    // // // print<<<1,1>>>(dec);
    

    // //     for(int i = 0; i < 10; i++){
    // //         printf("%lf\n",plainb[i]);
    // //     }
    hipDeviceSynchronize();
}