
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cstdio>
#include <iostream>
#include"hip/hip_runtime_api.h"

#define NX 1024
#define BATCH 1
using namespace std;
#define Check(status)\
{\
	if (status != hipSuccess)\
	{\
		cout << "行號:" << __LINE__ << endl;\
		cout << "錯誤:" << hipGetErrorString(status) << endl;\
	}\
}
int main(){
    hipfftComplex *data_h;
    hipfftComplex *data;
    Check(hipHostMalloc(&data_h,NX*sizeof(hipfftComplex), hipHostMallocDefault, hipHostMallocDefault));
    Check(hipMalloc(&data,NX * sizeof(hipfftComplex)));
    hipfftComplex *res = (hipfftComplex *)malloc(NX * sizeof(hipfftComplex));
    for(int i =0; i < NX/2; i++){
        data_h[i].x = (i * 113 + 71) % 5 * 1.41;
        data_h[i].y = 0;    
    }
    for(int i =0; i < NX/2; i++){
        data_h[NX-i-1].x = (i * 113 + 71) % 5 * 1.41;
        data_h[NX-i-1].y = 0;
    }
    hipMemcpy(data,data_h,NX * sizeof(hipfftComplex),hipMemcpyHostToDevice);
    hipfftHandle plan;
    hipfftPlan1d(&plan, NX, HIPFFT_C2C,BATCH);

    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

    hipfftDestroy(plan);
    Check(hipMemcpy(res,data,NX * sizeof(hipfftComplex),hipMemcpyDeviceToHost));
    hipFree(data);
    for(int i = 0;i < 10;i++){
        printf("%lf,%lf\n",res[i].x,res[i].y);
    }
    return 0;
}
