
#include <hip/hip_runtime.h>
#include"iostream"
#include"hip/hip_runtime_api.h"

#include"hipfft/hipfft.h"
using namespace std;
//FFT反變換後，用於規範化的函數
__global__ void normalizing(hipfftDoubleComplex* data,int data_len)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	data[idx].x /= data_len;
	data[idx].y /= data_len;
}
void Check(hipError_t status)
{
	if (status != hipSuccess)
	{
		cout << "行號:" << __LINE__ << endl;
		cout << "錯誤:" << hipGetErrorString(status) << endl;
	}
}
int main()
{
	const int Nt =4;
	const int BATCH = 1;
	//BATCH用於批量處理一批一維數據，當BATCH=2時
	//則將0-1024，1024-2048作爲兩個一維信號做FFT處理變換
	hipfftDoubleComplex* host_in, *host_out, *device_in, *device_out;
	//主機內存申請及初始化--主機鎖頁內存
	Check(hipHostMalloc((void**)&host_in, Nt * sizeof(hipfftDoubleComplex), hipHostMallocDefault));
	Check(hipHostMalloc((void**)&host_out, Nt * sizeof(hipfftDoubleComplex), hipHostMallocDefault));
	// for (int i = 1; i < Nt; i+=2)
	// {
	// 	host_in[i].x = 1;
	// 	host_in[i].y = 100;
	// }

	// for (int i = 0; i < Nt; i+=1)
	// {
	// 	host_in[i].x = 1;
	// 	// host_in[i].y = -100;
	// }
	host_in[0].x = 1;host_in[0].y = 1;
	host_in[1].x = 3;host_in[1].y = -4;
	host_in[2].x = 3;host_in[2].y=4;
	host_in[3].x = 1;host_in[3].y = -1;

	//設備內存申請
	Check(hipMalloc((void**)&device_in, Nt * sizeof(hipfftDoubleComplex)));
	Check(hipMalloc((void**)&device_out, Nt * sizeof(hipfftDoubleComplex)));
	//數據傳輸--H2D
	Check(hipMemcpy(device_in, host_in, Nt * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));

	//創建cufft句柄
	hipfftHandle cufftForwrdHandle, cufftInverseHandle;
	hipfftPlan1d(&cufftForwrdHandle, Nt, HIPFFT_Z2Z, BATCH);
	hipfftPlan1d(&cufftInverseHandle, Nt, HIPFFT_Z2Z, BATCH);

	//執行fft正變換
	hipfftExecZ2Z(cufftForwrdHandle, device_in, device_out, HIPFFT_FORWARD);

	//數據傳輸--D2H
	Check(hipMemcpy(host_out, device_out, Nt * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));

	//設置輸出精度--正變換結果輸出
	cout << "逆變換結果:" << endl;
	// cout.setf(20);
	for (int i = 0; i < Nt; i++)
	{
		cout << host_out[i].x<< "+j*" << host_out[i].y << endl;
	}

	//執行fft反變換
	// cufftExecZ2Z(cufftInverseHandle,  device_out, device_in, CUFFT_INVERSE);
	
	// //IFFT結果是真值的N倍，因此要做/N處理
	// dim3 grid(Nt/128); 
	// dim3 block(128);
	// normalizing << <grid, block >> > (device_in,Nt);

	// //數據傳輸--D2H
	// Check(cudaMemcpy(host_in, device_in, Nt * sizeof(cufftDoubleComplex), cudaMemcpyDeviceToHost));

	// //設置輸出精度--反變換結果輸出
	// cout << "反變換結果:" << endl;
	// // cout.setf(20);
	// for (int i = 0; i < Nt; i++)
	// {
	// 	cout << host_in[i].x << "+j*" << host_in[i].y << endl;
	// }
	// // cin.get();
	return 0;
}