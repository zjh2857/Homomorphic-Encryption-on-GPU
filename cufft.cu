#include "hip/hip_runtime.h"
#include"iostream"
#include"hip/hip_runtime_api.h"
#include""
#include"hipfft/hipfft.h"
using namespace std;
#define Check(call)														\
{																		\
	hipError_t status = call;											\
	if (status != hipSuccess)											\
	{																	\
		cout << "行號:" << __LINE__ << endl;							\
		cout << "錯誤:" << hipGetErrorString(status) << endl;			\
	}																	\
}

//FFT反變換後，用於規範化的函數
__global__ void normalizing(hipfftDoubleReal* data, int data_len)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	if (idx<data_len)
	{
		data[idx] /=(data_len);
	}
}

int main()
{
	const int Nt = 4;
	const int BATCH = 1;
	//BATCH用於批量處理一批一維數據，當BATCH=2時
	//則將0-1024，1024-2048作爲兩個一維信號做FFT處理變換
	hipfftDoubleReal* host_in,  *device_in;
	hipfftDoubleComplex* host_out, *device_out;
	//主機內存申請及初始化--主機鎖頁內存
	Check(hipHostMalloc((void**)&host_in, Nt * sizeof(hipfftDoubleReal)));
	//特別要注意：這裏的輸出長度變爲(Nt/2+1)
	Check(hipHostMalloc((void**)&host_out, (Nt / 2 + 1) * sizeof(hipfftDoubleComplex)));
	for (int i = 0; i < Nt; i++)
	{
		host_in[i] = 1;
	}
	host_in[1] = -1;
	//設備內存申請
	Check(hipMalloc((void**)&device_in, Nt * sizeof(hipfftDoubleReal)));
	//特別要注意：這裏的輸出長度變爲(Nt/2+1)
	Check(hipMalloc((void**)&device_out, (Nt / 2 + 1) * sizeof(hipfftDoubleComplex)));
	
	//數據傳輸--H2D
	Check(hipMemcpy(device_in, host_in, Nt * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice));

	//創建cufft句柄
	hipfftHandle cufftForwrdHandle, cufftInverseHandle;
	hipfftPlan1d(&cufftForwrdHandle, Nt, HIPFFT_D2Z, BATCH);
	hipfftPlan1d(&cufftInverseHandle, Nt, HIPFFT_Z2D, BATCH);

	//執行fft正變換
	hipfftExecD2Z(cufftForwrdHandle, device_in, device_out);//由於D2Z的方向是固定的，無需填入參數

	//數據傳輸--D2H
	Check(hipMemcpy(host_out, device_out, (Nt/2+1) * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));

	//設置輸出精度--正變換結果輸出
	cout << "正變換結果:" << endl;
	// cout.setf(20);
	for (int i = 0; i < (Nt / 2 + 1); i++)
	{
		cout << host_out[i].x << "+j*" << host_out[i].y << endl;
	}

	//執行fft反變換
	hipfftExecZ2D(cufftInverseHandle, device_out, device_in);//由於Z2D的方向是固定的，無需填入參數
	
	//IFFT結果是真值的N倍，因此要做/N處理
	dim3 grid(ceil((Nt / 2 + 1) / 128.0) + 1);
	dim3 block(128);
	normalizing << <grid, block >> > (device_in, Nt);

	//數據傳輸--D2H
	Check(hipMemcpy(host_in, device_in, Nt * sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost));

	//設置輸出精度--反變換結果輸出
	cout << "反變換結果:" << endl;
	// cout.setf(20);
	for (int i = 0; i < Nt; i++)
	{
		cout << host_in[i] << endl;
	}
	// cin.get();
	return 0;
}