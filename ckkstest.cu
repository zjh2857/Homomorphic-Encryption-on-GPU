#include "hip/hip_runtime.h"
#include "encoder.cuh"
#include"hipfft/hipfft.h"
#include "encryptor.cuh"
#include "evaluator.cuh"

__global__ void print(unsigned long long* a){
    for(int i = 0; i < 8;i++){
        printf("%llu,",a[i]);
    }printf("\n");
}
__global__ void print(unsigned long long* a,unsigned long long id){
    printf("%llu\n",id);
    for(int i = 0; i < 8;i++){
        printf("%llu ",a[i]);
    }printf("\n");
}
__global__ void print_d(unsigned long long* a,int d){
    for(int i = 0; i < 1;i++){
        printf("printf_d%d:%llu ",d,a[d]);
    }printf("\n");
}
int main(){
    printf("===\n");
    int N = 2048;
    double scale = 1llu << 30;
    double a[N];
    for(int i = 0; i < N; i++){
        a[i] = i ;
    }
    // a[N/2-1] = 10000;
    // a[0] = 10;
    double b[N];
    for(int i = 0; i < N; i++){
        b[i] = 2;
    }

    keyGen keygen(N,scale,8);
    Encoder encoder(N,scale,8);
    Encryptor encryptor(N,scale,8);
    Evaluator evaluator(N,8);
    auto encodeVeca = encoder.encode(a);
    // evaluator.rescale(encodeVeca);
    auto encodeVecb = encoder.encode(b);
    // // print<<<1,1>>>(encodeVeca);
    // // print<<<1,1>>>(encodeVecb);
    // // hipDeviceSynchronize();


    auto ciptertexta = encryptor.encrypt(encodeVeca,keygen.pub);
    // evaluator.rescale(ciptertexta);
    // // print<<<1,1>>>(keygen.pub.a);
    // // print<<<1,1>>>(keygen.pub.b);
    auto ciptertextb = encryptor.encrypt(encodeVecb,keygen.pub);
    // // printf("%p,%p\n",ciptertexta,ciptertextb);
    // //     print<<<1,1>>>(ciptertexta.a);
    // // print<<<1,1>>>(ciptertexta.b);
    // // print<<<1,1>>>(ciptertextb.a);
    // // print<<<1,1>>>(ciptertextb.b);
    // // auto plaina = encoder.decode(encodeVeca);
    // print<<<1,1>>>(encodeVecb);

    auto ciptertextc = evaluator.mulcipter(ciptertexta,ciptertextb);

    // auto ciptertextd = evaluator.relien(ciptertextc,keygen.relien);
    // print<<<1,1>>>(ciptertextd.a);
    // print<<<1,1>>>(ciptertextd.b);
    // print<<<1,1>>>(ciptertextd.c);
    // evaluator.mulPlain(ciptertexta,encodeVecb);
    // // evaluator.mulPlain(encodeVeca,encodeVecb);

    // // printf("###\n");
    // // print<<<1,1>>>(ciptertexta.a);
    // // print<<<1,1>>>(ciptertexta.b);    
    // encryptor.decrypt(ciptertextb,keygen.pri);
    unsigned long long* dec = encryptor.decrypt(ciptertextc,keygen.pri);
    print<<<1,1>>>(dec);
    // // print<<<1,1>>>(dec);
    auto plaina = encoder.decode(dec);
    // auto plaina = encoder.decode(encodeVeca);
    // // // auto plainb =  encoder.decode(encodeVecb);
    // // // print<<<1,1>>>(dec);
    

    for(int i = 0; i < 20; i++){
        printf("%lf\n",plaina[i]/scale);
    }
    //  auto plainb = encoder.decode(encodeVecb);
    // // // print<<<1,1>>>(dec);
    

    // //     for(int i = 0; i < 10; i++){
    // //         printf("%lf\n",plainb[i]);
    // //     }
    hipDeviceSynchronize();
}