#include "hip/hip_runtime.h"
#include "encoder.cuh"
#include"hipfft/hipfft.h"
#include "encryptor.cuh"
#include "evaluator.cuh"

__global__ void print(unsigned long long* a){
    for(int i = 0; i < 8;i++){
        printf("%llu ",a[i]);
    }printf("\n");
}
__global__ void print_d(unsigned long long* a,int d){
    for(int i = 0; i < 1;i++){
        printf("printf_d%d:%llu ",d,a[d]);
    }printf("\n");
}
int main(){
    int N = 2048;
    double scale = 1 << 20;
    double a[N];
    for(int i = 0; i < N; i++){
        a[i] = i;
    }
    double b[N];
    for(int i = 0; i < N; i++){
        b[i] = 0;
    }
    b[0] = 100;
    keyGen keygen(N,scale);
    Encoder encoder(N,scale,8);
    // Encryptor encryptor(N,scale);
    Evaluator evaluator(N);

    auto encodeVeca = encoder.encode(a);
    auto encodeVecb = encoder.encode(b);
    // print<<<1,1>>>(encodeVeca);


    // auto ciptertexta = encryptor.encrypt(encodeVeca,keygen.pub);
    // // print<<<1,1>>>(ciptertexta.a);
    // // print<<<1,1>>>(ciptertexta.b);

    // auto ciptertextb = encryptor.encrypt(encodeVecb,keygen.pub);
    // printf("%p,%p\n",ciptertexta,ciptertextb);
    //     print<<<1,1>>>(ciptertexta.a);
    // print<<<1,1>>>(ciptertexta.b);
    // print<<<1,1>>>(ciptertextb.a);
    // print<<<1,1>>>(ciptertextb.b);
    // auto plaina = encoder.decode(encodeVeca);
    evaluator.mulPlain(encodeVeca,encodeVecb);
    
    // print<<<1,1>>>(ciptertexta.a);
    // print<<<1,1>>>(ciptertexta.b);    
    // encryptor.decrypt(ciptertextb,keygen.pri);
    // unsigned long long* dec = encryptor.decrypt(ciptertexta,keygen.pri);
    auto plaina = encoder.decode(encodeVeca);
    // print<<<1,1>>>(dec);
    

        for(int i = 0; i < 10; i++){
            printf("%lf\n",plaina[i]);
        }
    //  auto plainb = encoder.decode(encodeVecb);
    // // print<<<1,1>>>(dec);
    

    //     for(int i = 0; i < 10; i++){
    //         printf("%lf\n",plainb[i]);
    //     }
    hipDeviceSynchronize();
}