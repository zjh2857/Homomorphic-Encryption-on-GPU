#include "hip/hip_runtime.h"
#include "encoder.cuh"
#include"hipfft/hipfft.h"
// __global__ void print(unsigned long long* a){
//     printf("%llu\n",a[0]);
// }
int main(){
    int N = 4096;
    double a[N];
    for(int i = 0; i < N; i++){
        a[i] = 1;
    }


    Encoder encoder(N,100);
    auto encodeVec = encoder.encode(a);
    // print<<<1,1>>>(d_a);
    auto plain = encoder.decode(encodeVec);
    // print<<<1,1>>>(d_a);
    for(int i = 0; i < 10; i++){
        printf("%lf\n",plain[i]);
    }
    hipDeviceSynchronize();
}