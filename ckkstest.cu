#include "hip/hip_runtime.h"
#include "encoder.cuh"
#include"hipfft/hipfft.h"
#include "encryptor.cuh"
#include "evaluator.cuh"
__global__ void print(unsigned long long* a){
    for(int i = 0; i < 8;i++){
        printf("%llu ",a[i]);
    }printf("\n");
}
int main(){
    int N = 4096;
    double scale = 100000;
    double a[N];
    for(int i = 0; i < N; i++){
        a[i] = 1000;
    }
    double b[N];
    for(int i = 0; i < N; i++){
        b[i] = 1000;
    }
// a[0] = 2;
    keyGen keygen(N,scale);
    Encoder encoder(N,scale);
    Encryptor encryptor(N,scale);
    Evaluator evaluator(N);

    auto encodeVeca = encoder.encode(a);
    auto encodeVecb = encoder.encode(b);
    // print<<<1,1>>>(encodeVeca);


    auto ciptertexta = encryptor.encrypt(encodeVeca,keygen.pub);
    // print<<<1,1>>>(ciptertexta.a);
    // print<<<1,1>>>(ciptertexta.b);

    auto ciptertextb = encryptor.encrypt(encodeVecb,keygen.pub);
    // printf("%p,%p\n",ciptertexta,ciptertextb);
    //     print<<<1,1>>>(ciptertexta.a);
    // print<<<1,1>>>(ciptertexta.b);
    // print<<<1,1>>>(ciptertextb.a);
    // print<<<1,1>>>(ciptertextb.b);
    evaluator.addcipter(ciptertexta,ciptertextb);
    
    // print<<<1,1>>>(ciptertexta.a);
    // print<<<1,1>>>(ciptertexta.b);    
    // encryptor.decrypt(ciptertextb,keygen.pri);
    unsigned long long* dec = encryptor.decrypt(ciptertexta,keygen.pri);
    auto plain = encoder.decode(dec);
    // print<<<1,1>>>(dec);
    

        for(int i = 0; i < 10; i++){
            printf("%lf\n",plain[i]);
        }
 
        hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        // Possibly: exit(-1) if program cannot continue....
    }
}