#include "hip/hip_runtime.h"
#include "encoder.cuh"
#include"hipfft/hipfft.h"
#include "encryptor.cuh"
__global__ void print(unsigned long long* a){
    for(int i = 0; i < 8;i++){
        printf("%llu ",a[i]);
    }printf("\n");
}
int main(){
    int N = 4096;
    double scale = 10000;
    double a[N];
    for(int i = 0; i < N; i++){
        a[i] = 20000;
    }

// a[0] = 2;
    keyGen keygen(N,scale);
    Encoder encoder(N,scale);
    auto encodeVec = encoder.encode(a);

// print<<<1,1>>>(encodeVec);

    Encryptor encryptor(N,scale);
    auto ciptertext = encryptor.encrypt(encodeVec,keygen.pub);
    unsigned long long* dec = encryptor.decrypt(ciptertext,keygen.pri);
    auto plain = encoder.decode(dec);
    print<<<1,1>>>(dec);


        for(int i = 0; i < 10; i++){
            printf("%lf\n",plain[i]);
        }
 
                hipDeviceSynchronize();
hipError_t err = hipGetLastError();
if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
    // Possibly: exit(-1) if program cannot continue....
}
}