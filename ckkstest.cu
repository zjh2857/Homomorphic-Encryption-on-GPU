#include "hip/hip_runtime.h"
#include "encoder.cuh"
#include"hipfft/hipfft.h"
#include "encryptor.cuh"
#include "evaluator.cuh"

__global__ void print(unsigned long long* a){
    for(int i = 0; i < 8;i++){
        printf("%llu,",a[i]);
    }printf("\n");
}
__global__ void print(unsigned long long* a,unsigned long long id){
    printf("%llu\n",id);
    for(int i = 0; i < 8;i++){
        printf("%llu ",a[i]);
    }printf("\n");
}
__global__ void print_d(unsigned long long* a,int d){
    for(int i = 0; i < 1;i++){
        printf("printf_d%d:%llu ",d,a[d]);
    }printf("\n");
}
int main(int argc,char *argv[]){
    printf("===\n");
    int N = 2048;
    double scale = 1llu << 30;
    double a[N];
    for(int i = 0; i < N; i++){
        // if(i&1){
        //     a[i] = 1;
        // }
        // else{
        //     a[i] = -1;
        // }
        a[i] = N - i - 1;
    }
    // a[N/2-1] = 10000;
    // a[0] = 10;
    double b[N];
    for(int i = 0; i < N; i++){
        b[i] = i;
    }
    // b[N-1] = a[0];

    keyGen keygen(N,scale,8);
    Encoder encoder(N,scale,8);
    Encryptor encryptor(N,scale,8);
    Evaluator evaluator(N,8);
    // encoder.test();
    auto encodeVeca = encoder.encode(a);
    // auto ciptertexta = encryptor.encrypt(encodeVeca,keygen.pub);

    // int t = atoi(argv[1]);
    // for(int i = 0; i < t;i++){
    //     evaluator.rotation(encodeVeca);
    // }
    // auto ciptertextr = evaluator.sum(ciptertexta,keygen.galois);
    // evaluator.rotation(encodeVeca);
    // for(int i = 0;  i < N; i++){
    //     evaluator.rotation(encodeVeca);
    //     auto plaina = encoder.decode(encodeVeca);
    //     printf("%lf\n",plaina[0]);
    // }
    // evaluator.rotation(encodeVeca);

    // evaluator.rescale(encodeVeca);
    auto encodeVecb = encoder.encode(b);
    // // print<<<1,1>>>(encodeVeca);
    // // print<<<1,1>>>(encodeVecb);
    // // hipDeviceSynchronize();


    auto ciptertexta = encryptor.encrypt(encodeVeca,keygen.pub);
    // auto ciptertextr = evaluator.sum(ciptertexta,keygen.galois);
    // evaluator.rescale(ciptertexta);
    // // print<<<1,1>>>(keygen.pub.a);
    // // print<<<1,1>>>(keygen.pub.b);
    auto ciptertextb = encryptor.encrypt(encodeVecb,keygen.pub);
    auto ciptertextr = evaluator.innerProduct(ciptertexta,ciptertextb,keygen.galois,keygen.relien);
    // // printf("%p,%p\n",ciptertexta,ciptertextb);
    // //     print<<<1,1>>>(ciptertexta.a);
    // // print<<<1,1>>>(ciptertexta.b);
    // // print<<<1,1>>>(ciptertextb.a);
    // // print<<<1,1>>>(ciptertextb.b);
    // // auto plaina = encoder.decode(encodeVeca);
    // print<<<1,1>>>(encodeVecb);

    // auto ciptertextc = evaluator.mulcipter(ciptertexta,ciptertextb);

    // auto ciptertextd = evaluator.relien(ciptertextc,keygen.relien);
    // evaluator.rescale(ciptertextd);
    // print<<<1,1>>>(ciptertextd.a);
    // print<<<1,1>>>(ciptertextd.b);
    // print<<<1,1>>>(ciptertextd.c);
    // evaluator.mulPlain(ciptertexta,encodeVecb);
    // // evaluator.mulPlain(encodeVeca,encodeVecb);

    // // printf("###\n");
    // // print<<<1,1>>>(ciptertexta.a);
    // // print<<<1,1>>>(ciptertexta.b);    
    // encryptor.decrypt(ciptertextb,keygen.pri);
    unsigned long long* dec = encryptor.decrypt(ciptertextr,keygen.pri);
    // print<<<1,1>>>(dec);
    // // print<<<1,1>>>(dec);
    auto plaina = encoder.decode(dec);
    // auto plaina = encoder.decode(encodeVecd);
    // // // auto plainb =  encoder.decode(encodeVecb);
    // // // print<<<1,1>>>(dec);
    
    // printf("%lf",plaina[0]);
    for(int i = 0; i < 10; i++){
        printf("%d,%lf\n",i,plaina[i]/scale);
    }
    //  auto plainb = encoder.decode(encodeVecb);
    // // // print<<<1,1>>>(dec);
    

    // //     for(int i = 0; i < 10; i++){
    // //         printf("%lf\n",plainb[i]);
    // //     }
    hipDeviceSynchronize();
}