#include "hip/hip_runtime.h"
#include "encoder.cuh"
#include"hipfft/hipfft.h"
#include "encryptor.cuh"
#include "evaluator.cuh"
__global__ void print(unsigned long long* a){
    for(int i = 0; i < 8;i++){
        printf("%llu ",a[i]);
    }printf("\n");
}
int main(){
    int N = 2048;
    double scale = 1000;
    double a[N];
    for(int i = 0; i < N; i++){
        a[i] = 2;
    }
    double b[N];
    for(int i = 0; i < N; i++){
        b[i] = 4;
    }
// a[0] = 2;
    keyGen keygen(N,scale);
    Encoder encoder(N,scale);
    Encryptor encryptor(N,scale);
    Evaluator evaluator(N);

    auto encodeVeca = encoder.encode(a);
    auto encodeVecb = encoder.encode(b);
    // print<<<1,1>>>(encodeVeca);


    // auto ciptertexta = encryptor.encrypt(encodeVeca,keygen.pub);
    // // print<<<1,1>>>(ciptertexta.a);
    // // print<<<1,1>>>(ciptertexta.b);

    // auto ciptertextb = encryptor.encrypt(encodeVecb,keygen.pub);
    // printf("%p,%p\n",ciptertexta,ciptertextb);
    //     print<<<1,1>>>(ciptertexta.a);
    // print<<<1,1>>>(ciptertexta.b);
    // print<<<1,1>>>(ciptertextb.a);
    // print<<<1,1>>>(ciptertextb.b);
    // evaluator.mulPlain(encodeVeca,encodeVecb);
    
    // print<<<1,1>>>(ciptertexta.a);
    // print<<<1,1>>>(ciptertexta.b);    
    // encryptor.decrypt(ciptertextb,keygen.pri);
    // unsigned long long* dec = encryptor.decrypt(ciptertexta,keygen.pri);
    auto plaina = encoder.decode(encodeVeca);
    // print<<<1,1>>>(dec);
    

        for(int i = 0; i < 10; i++){
            printf("%lf\n",plaina[i]);
        }
     auto plainb = encoder.decode(encodeVecb);
    // print<<<1,1>>>(dec);
    

        for(int i = 0; i < 10; i++){
            printf("%lf\n",plainb[i]);
        }
}