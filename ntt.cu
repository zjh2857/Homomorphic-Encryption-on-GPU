
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

const long long M = 998244353;
const long long N = 8;
long long l = 0;
long long Ninv;
long long revphi[N];
long long revphiinv[N];
long long bitrevs[N];
long long bitrevl[N];

void print(long long *a){
    for(int i = 0; i < N; i++){
        cout << a[i] << " ";
    }
    cout << endl;    
}

long long qpow(long long a,long long b,long long n){
    long long ans = 1;
    long long base = a;
    while(b){
        if(b&1){
            ans *= base;
            ans %= n;
        }
        base = base * base % n;
        b >>= 1;
    }
    return ans;
}

long long _bitrevs(long long x){
    long long res = 0;
    for(int i = 0; i < l - 1; i++){
        res = (res << 1) | (x & 1);
        x >>= 1;
    }
    return res;
}
long long _bitrevl(long long x){
    long long res = 0;

    for(int i = 0; i < l; i++){
        res = (res << 1) | (x & 1);
        x >>= 1;
    }
    return res;
}
void init(){
    long long temp = N;
    while(temp != 1){
        temp=temp>>1;
        l++;
    }
    long long g = qpow(3,(M-1)/N,M);
    long long gi = qpow(332748118,(M-1)/N,M);
    for(int i = 0; i < N; i++){
        bitrevs[i] = _bitrevs(i);
        bitrevl[i] = _bitrevl(i);
    }
    for(int i = 0; i < N/2; i++){
        revphi[i] = qpow(g,bitrevs[i],M);
        revphiinv[i] = qpow(gi,bitrevs[i],M);
    }
    Ninv = qpow(N,M-2,M);
    
}
void swapInv(long long* a){
    for(int i = 0; i < N; i++){
        if(i < bitrevl[i])swap(a[i],a[bitrevl[i]]);
    }
}
long long* NTT(long long* a){
    long long t = N;
    long long m = 1;
    while(t > 1){
        t /= 2;
        for(int i = 0; i < m; i++){
            int j1 = 2 * i * t;
            int j2 = j1 + t;
            for(int j = j1; j < j2; j++){
                long long u = a[j] % M;
                long long v = a[j + t] * revphi[i] % M;
                a[j] = (u + v + M) % M;
                a[j + t] = (u - v + M) % M;
            }
        }
        m <<= 1;
    }
    return a;
}

long long* INTT(long long* a){
    swapInv(a);
    long long t = N;
    long long m = 1;
    while(t > 1){
        t /= 2;
        for(int i = 0; i < m; i++){
            int j1 = 2 * i * t;
            int j2 = j1 + t;
            for(int j = j1; j < j2; j++){
                long long u = a[j] % M;
                long long v = a[j + t] * revphiinv[i] % M;
                a[j] = (u + v + M) % M;
                a[j + t] = (u - v + M) % M;
            }
        }

        m <<= 1;
    }
    for(int i = 0; i < N; i++){
        a[i] = a[i] * Ninv % M;
    }
    swapInv(a);
    return a;
}

__global__ void cuNTT(long long* a,long long* revphi,long long l){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i < l; i++){
        int t = (N/2) >> i;
        int address = idx / t * t + idx;
        long long u = a[address];
        long long v = a[address+t] * revphi[idx/t] % M;

        a[address] = (u + v + M) % M;
        a[address + t] = (u - v + M) % M;
        __syncthreads();
    }
}


__global__ void cuINTT(long long* a,long long* revphiinv,long long l){
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i < l; i++){
        int t = (N/2) >> i;
        int address = idx / t * t + idx;
        long long u = a[address];
        long long v = a[address+t] * revphiinv[idx/t] % M;

        a[address] = (u + v + M) % M;
        a[address + t] = (u - v + M) % M;
        __syncthreads();
    }
}

__global__ void cuSwapInv(long long* a,long long* bitrevl){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < bitrevl[idx]){
        long long t = a[idx];
        a[idx] = a[bitrevl[idx]];
        a[bitrevl[idx]] = t;
    }
}

int main(){
    long long* a = (long long*)malloc(N * sizeof(long long));
    for(int i = 0; i < N;i++){
        a[i] = (i * 141 + 54) % 17;
    }
    init();
    int nByte = N * sizeof(long long);
    long long *a_d,*revphi_d,*revphiinv_d,*bitrevl_d;
    hipMalloc(&a_d,nByte);
    hipMemcpy(a_d,a,nByte,hipMemcpyHostToDevice);
    hipMalloc(&revphi_d,nByte);
    hipMemcpy(revphi_d,revphi,nByte,hipMemcpyHostToDevice);
    hipMalloc(&revphiinv_d,nByte);
    hipMemcpy(revphiinv_d,revphiinv,nByte,hipMemcpyHostToDevice);
    hipMalloc(&bitrevl_d,nByte);
    hipMemcpy(bitrevl_d,bitrevl,nByte,hipMemcpyHostToDevice);
    print(a);
    NTT(a);
    print(a);
    INTT(a);
    print(a);
    
    dim3 block_half(N/2);
    dim3 grid_half(N/2/block_half.x);
    dim3 block(N);
    dim3 grid(N/block.x); 
    long long* res = (long long *)malloc(nByte);
    cuNTT<<<grid_half,block_half>>>(a_d,revphi_d,l);
    hipMemcpy(res,a_d,nByte,hipMemcpyDeviceToHost);
    print(res);
    cuSwapInv<<<grid,block>>>(a_d,bitrevl_d);
    hipMemcpy(res,a_d,nByte,hipMemcpyDeviceToHost);
    print(res);
    cuINTT<<<grid_half,block_half>>>(a_d,revphiinv_d,l);
    hipMemcpy(res,a_d,nByte,hipMemcpyDeviceToHost);
    print(res);
    cuSwapInv<<<grid,block>>>(a_d,bitrevl_d);
    hipMemcpy(res,a_d,nByte,hipMemcpyDeviceToHost);
    print(res);
}