#include "hip/hip_runtime.h"
/*

　　 _,,....,,_　 ＿人人人人人人人人人人人人人人人人人人人人人＿
-''":::::::::::::｀''＞　　　ゆっくりしていってね！！！　　　＜
ヽ:::::::::::::::::::::￣^Ｙ^Ｙ^Ｙ^Ｙ^Ｙ^Ｙ^Ｙ^Ｙ^Ｙ^Ｙ^Ｙ^￣
|::::::;ノ´￣＼:::::::::::＼_,. -‐ｧ　＿_　　 _____　　 ＿_____
|::::ﾉ　　　ヽ､ヽr-r'"´　　（.__　　　,´　_,, '-´￣￣｀-ゝ 、_ イ、
_,.!イ_　　_,.ﾍｰｧ'二ﾊ二ヽ､へ,_7　　'r ´　　　　　　　　　　ヽ、ﾝ、
::::::rｰ''7ｺ-‐'"´　 　 ;　 ',　｀ヽ/｀7　,'＝=─-　　　 　 -─=＝',　i
r-'ｧ'"´/　 /!　ﾊ 　ハ　 !　　iヾ_ﾉ　i　ｲ　iゝ、ｲ人レ／_ルヽｲ i　|
!イ´ ,' |　/__,.!/　V　､!__ﾊ　 ,'　,ゝﾚﾘｲi (ﾋ_] 　　 　ﾋ_ﾝ ).| .|、i .||
`! 　!/ﾚi'　(ﾋ_] 　　 　ﾋ_ﾝ ﾚ'i　ﾉ　　　!Y!""　 ,＿__, 　 "" 「 !ﾉ i　|
,'　 ﾉ 　 !'"　 　 ,＿__,　 "' i .ﾚ'　L.',.　 　ヽ _ﾝ　　　　L」 ﾉ| .|
　（　　,ﾊ　　　　ヽ _ﾝ　 　人! 　　　　| ||ヽ、　　　　　　 ,ｲ| ||ｲ| /
,.ﾍ,）､　　）＞,､ _____,　,.イ　 ハ　　　レ ル｀ ー--─ ´ルﾚ　ﾚ´

*/
#include <iostream>
#include "freshman.h"

using namespace std;

const long long M = 998244353;
const long long N = 1024;
long long l = 0;
long long Ninv;
long long revphi[N];
long long revphiinv[N];
long long bitrevs[N];
long long bitrevl[N];

void print(long long *a){
    for(int i = 0; i < N; i++){
        cout << a[i] << " ";
    }
    cout << endl;    
}

long long qpow(long long a,long long b,long long n){
    long long ans = 1;
    long long base = a;
    while(b){
        if(b&1){
            ans *= base;
            ans %= n;
        }
        base = base * base % n;
        b >>= 1;
    }
    return ans;
}

long long _bitrevs(long long x){
    long long res = 0;
    for(int i = 0; i < l - 1; i++){
        res = (res << 1) | (x & 1);
        x >>= 1;
    }
    return res;
}
long long _bitrevl(long long x){
    long long res = 0;

    for(int i = 0; i < l; i++){
        res = (res << 1) | (x & 1);
        x >>= 1;
    }
    return res;
}
void init(){
    long long temp = N;
    while(temp != 1){
        temp=temp>>1;
        l++;
    }
    long long g = qpow(3,(M-1)/N,M);
    long long gi = qpow(332748118,(M-1)/N,M);
    for(int i = 0; i < N; i++){
        bitrevs[i] = _bitrevs(i);
        bitrevl[i] = _bitrevl(i);
    }
    for(int i = 0; i < N/2; i++){
        revphi[i] = qpow(g,bitrevs[i],M);
        revphiinv[i] = qpow(gi,bitrevs[i],M);
    }
    Ninv = qpow(N,M-2,M);
    
}
void swapInv(long long* a){
    for(int i = 0; i < N; i++){
        if(i < bitrevl[i])swap(a[i],a[bitrevl[i]]);
    }
}
long long* NTT(long long* a){
    long long t = N;
    long long m = 1;
    while(t > 1){
        t /= 2;
        for(int i = 0; i < m; i++){
            int j1 = 2 * i * t;
            int j2 = j1 + t;
            for(int j = j1; j < j2; j++){
                long long u = a[j] % M;
                long long v = a[j + t] * revphi[i] % M;
                a[j] = (u + v + M) % M;
                a[j + t] = (u - v + M) % M;
            }
        }
        m <<= 1;
    }
    return a;
}

long long* INTT(long long* a){
    swapInv(a);
    long long t = N;
    long long m = 1;
    while(t > 1){
        t /= 2;
        for(int i = 0; i < m; i++){
            int j1 = 2 * i * t;
            int j2 = j1 + t;
            for(int j = j1; j < j2; j++){
                long long u = a[j] % M;
                long long v = a[j + t] * revphiinv[i] % M;
                a[j] = (u + v + M) % M;
                a[j + t] = (u - v + M) % M;
            }
        }

        m <<= 1;
    }
    for(int i = 0; i < N; i++){
        a[i] = a[i] * Ninv % M;
    }
    swapInv(a);
    return a;
}

__global__ void cuNTT(long long* a,long long *b,long long* revphi,long long l){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i < l; i++){
        int t = (N/2) >> i;
        int address = idx / t * t + idx;
        long long u = a[address];
        long long v = a[address+t] * revphi[idx/t] % M;

        a[address] = (u + v + M) % M;
        a[address + t] = (u - v + M) % M;
        __syncthreads();
    }
}

__global__ void cuNttSuffle(long long* a,long long *b, long long* revphi,long long l){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long val = a[idx];
    
    for(int i = 0; i < l; i++){
        int t = (N/2) >> i;
        long long r = revphi[idx/t/2];
        bool b = idx & t;
        long long getVal = __shfl_xor(val,t,32);
        // long long v = (r * getVal) % M;
        if(b){
            val = (getVal - (val * r)%M + 2 * M) % M;
        }else{
            val = (val + (getVal * r)%M + 2 * M) % M;
        }
        __syncthreads();
    }
    b[idx] = val;
}
__global__ void cuINTT(long long* a,long long* revphiinv,long long l){
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i < l; i++){
        int t = (N/2) >> i;
        int address = idx / t * t + idx;
        long long u = a[address];
        long long v = a[address+t] * revphiinv[idx/t] % M;

        a[address] = (u + v + M) % M;
        a[address + t] = (u - v + M) % M;
        __syncthreads();
    }
}

__global__ void cuSwapInv(long long* a,long long* bitrevl){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < bitrevl[idx]){
        long long t = a[idx];
        a[idx] = a[bitrevl[idx]];
        a[bitrevl[idx]] = t;
    }
}

int main(){
    long long* a = (long long*)malloc(N * sizeof(long long));
    for(int i = 0; i < N;i++){
        a[i] = (i * 141 + 54) % 17;
    }
    init();
    int nByte = N * sizeof(long long);
    long long *a_d,*b_d,*revphi_d,*revphiinv_d,*bitrevl_d;
    hipMalloc(&a_d,nByte);
    hipMalloc(&b_d,nByte);
    hipMemcpy(a_d,a,nByte,hipMemcpyHostToDevice);
    hipMalloc(&revphi_d,nByte);
    hipMemcpy(revphi_d,revphi,nByte,hipMemcpyHostToDevice);
    hipMalloc(&revphiinv_d,nByte);
    hipMemcpy(revphiinv_d,revphiinv,nByte,hipMemcpyHostToDevice);
    hipMalloc(&bitrevl_d,nByte);
    hipMemcpy(bitrevl_d,bitrevl,nByte,hipMemcpyHostToDevice);
    print(a);
    NTT(a);
    print(a);
    INTT(a);
    print(a);
    
    dim3 block_half(N/2);
    dim3 grid_half(N/2/block_half.x);
    dim3 block(N);
    dim3 grid(N/block.x); 
    long long* res = (long long *)malloc(nByte);
    double start = cpuSecond();
    for(int i = 0; i < 100000;i++){
        cuNttSuffle<<<1,N>>>(a_d,b_d,revphi_d,l);
    } 
    hipMemcpy(res,b_d,nByte,hipMemcpyDeviceToHost);
    printf("Suffle time%lf\n",cpuSecond()-start);
    start = cpuSecond();
    for(int i = 0; i < 100000;i++){
        cuNTT<<<1,N/2>>>(a_d,b_d,revphi_d,l);
    } 
    hipMemcpy(res,a_d,nByte,hipMemcpyDeviceToHost);
    printf("cuNTT time%lf\n",cpuSecond() - start);
    // print(res);
    // cuNTT<<<grid_half,block_half>>>(a_d,revphi_d,l);
    // hipMemcpy(res,a_d,nByte,hipMemcpyDeviceToHost);
    // print(res);
    // cuSwapInv<<<grid,block>>>(a_d,bitrevl_d);
    // hipMemcpy(res,a_d,nByte,hipMemcpyDeviceToHost);
    // print(res);
    // cuINTT<<<grid_half,block_half>>>(a_d,revphiinv_d,l);
    // hipMemcpy(res,a_d,nByte,hipMemcpyDeviceToHost);
    // print(res);
    // cuSwapInv<<<grid,block>>>(a_d,bitrevl_d);
    // hipMemcpy(res,a_d,nByte,hipMemcpyDeviceToHost);
    // print(res);
}